#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
//#define xlns16_ideal
#define xlns16_alt 
#include "xlns16.cpp"
#include "xlns16d.cu"

#define ITER (37)
#define LNS_TESTS (1048576)
#define LNS_THREAD_CNT (32)
//#define LNS_CTA_CNT (256)
#define LNS_CTA_CNT (32768)

struct xlnsParams
{
 xlns16d_float *arglnsx;
 xlns16d_float *arglnsy;
 xlns16d_float *reslns;
 int n;
};



/*
 for (i=0; i<LNS_TESTS; i++)
 {
   res[i] = manxlnsiter(argx[i], argy[i], ITER); 
 }
*/


/*Mandelbrot set */

//__device__ xlns16d_float manxlnsiterd(xlns16d_float x, xlns16d_float y, int iter)
__device__ xlns16d_float manxlnsiterd(xlns16d_float x, xlns16d_float y)
{
	int count;
	xlns16d_float x1,y1,xnew,ynew,two,four,res;
	two = 2.0;
	four = 4.0;
	x1 = x;
	y1 = y;
	for (count=0; count<ITER; count++)
	{
		xnew = x*x - y*y + x1;
		ynew = x*y*two + y1;
		res = x*x+y*y;
		x = (res < four) ? xnew : x;
		y = (res < four) ? ynew : y;
	}
	return res;
}

__global__ void xlns_kernel(struct xlnsParams parms)
{
 int i;
 int totalThreads = gridDim.x * blockDim.x;
 int ctaStart = blockDim.x * blockIdx.x;
 for (i = ctaStart + threadIdx.x; i < parms.n; i += totalThreads)  
  {
    parms.reslns[i] = manxlnsiterd(parms.arglnsx[i],parms.arglnsy[i]);  //,ITER);
  }
}



int main (int argc, char *argv[])
{
 int i;
 //FILE* f;

 xlns16d_float * lnsRes = 0;
 xlns16d_float * lnsArgx = 0;
 xlns16d_float * lnsArgy = 0;
 xlns16_float * argx = 0;
 xlns16_float * argy = 0;
 xlns16_float * res = 0;
 xlns16_float zero;

 struct xlnsParams funcParams;
 int ix,iy;
 xlns16_float four,xscale,yscale;

 hipMalloc ((void **)&lnsArgx, LNS_TESTS * sizeof(xlns16d_float));
 hipMalloc ((void **)&lnsArgy, LNS_TESTS * sizeof(xlns16d_float));
 hipMalloc ((void **)&lnsRes, LNS_TESTS * sizeof(xlns16d_float));

 argx = (xlns16_float *) malloc (LNS_TESTS * sizeof(argx[0]));
 argy = (xlns16_float *) malloc (LNS_TESTS * sizeof(argy[0]));
 res = (xlns16_float *) malloc (LNS_TESTS * sizeof(res[0]));
 zero = 0;

 for (i=0; i<LNS_TESTS; i++)
 {
   argx[i] = zero;
   argy[i] = zero;
 }
 i=0;
 four = 4.0;
 yscale = 12.0;
 xscale = 24.0;
 for (iy = 11; iy >= -11; iy--)
	{
		for (ix=-40; ix <= 38; ix++)
		{
			argy[i] = ((float)iy)/yscale;
			argx[i] = ((float)ix)/xscale;
			i++;
		}
	}
 hipMemcpy (lnsArgx, argx, LNS_TESTS * sizeof(argx[0]), hipMemcpyHostToDevice);
 hipMemcpy (lnsArgy, argy, LNS_TESTS * sizeof(argy[0]), hipMemcpyHostToDevice);

 funcParams.arglnsx = lnsArgx;
 funcParams.arglnsy = lnsArgy;
 funcParams.reslns = lnsRes;
 funcParams.n = LNS_TESTS;
 
 printf("start CUDA\n");
 xlns_kernel<<<LNS_CTA_CNT,LNS_THREAD_CNT>>>(funcParams);
 printf("done CUDA\n");

 hipMemcpy (res, lnsRes, LNS_TESTS * sizeof(res[0]), hipMemcpyDeviceToHost);
 printf("done CUDA memcpy\n");

 i = 0;
 for (iy = 11; iy >= -11; iy--)
	{
		for (ix=-40; ix <= 38; ix++)
		{
			if (res[i] >= four)
				printf("*");
			else
				printf(" ");
			i++;
		}
		printf("\n");
	}


return 0;
}

